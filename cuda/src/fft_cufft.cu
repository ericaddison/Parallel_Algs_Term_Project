#include "fft_cuda.h"
#include <hipfft/hipfft.h>

void cuda_init()
{
	hipFree(0);
	return;
}

float fft_cufft(double *h_A, int n)
{

	// define device data
	hipfftDoubleComplex *d_A;
	hipMalloc((hipfftDoubleComplex**) &d_A, n*sizeof(hipfftDoubleComplex));
	hipMemcpy(d_A, h_A, n*sizeof(thCdouble), hipMemcpyHostToDevice);

	// make cufft plan
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_Z2Z , 1);

	// cuda timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// start timing
	hipEventRecord(start);
	

	// call cufft
	hipfftResult res = hipfftExecZ2Z(plan, d_A, d_A, HIPFFT_FORWARD);

	// stop timing
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	if(res != HIPFFT_SUCCESS)
		std::cout << "Warning: non-success cufft code found: " << res << std::endl;

	// copy back to host
    hipMemcpy(h_A, d_A, n*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipfftDestroy(plan);

	return milliseconds;
}

